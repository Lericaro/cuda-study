#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <common.h>
#include <cstdlib>

template <typename T> int check_diff(T *A, T *B, int n) {
    int res = 0;
    double eps = 1e-8;
    for (int i = 0; i < n; i++) {
        if (abs(A[i] - B[i]) > eps)
            ++res;
    }
    return res;
}

__global__ void sum_kernel(float *A, float *B, float *C, int nx, int ny) {
    int ix = threadIdx.x + blockDim.x * blockIdx.x;
    int iy = threadIdx.y + blockDim.y * blockIdx.y;

    int idx = ix + iy * nx;
    if (ix < nx && iy < ny) {
        C[idx] = A[idx] + B[idx];
    }
}

void sumMatrixHost(float *A, float *B, float *C, int nx, int ny) {
    for (int iy = 0; iy < ny; iy++) {
        for (int ix = 0; ix < nx; ix++) {
            int idx = iy * nx + ix;
            C[idx] = A[idx] + B[idx];
        }
    }
}

void sumMatrixDevice(float *A, float *B, float *C, int nx, int ny, int dx,
                     int dy) {
    dim3 block_dim(dx, dy);
    dim3 grid_dim((nx + dx - 1) / dx, (ny + dy - 1) / dy);
    sum_kernel<<<grid_dim, block_dim>>>(A, B, C, nx, ny);
}

int main() {
    srand(time(nullptr));
    int nx = 1 << 14, ny = 1 << 12;
    int nxy = nx * ny;
    int nbytes = nxy * sizeof(nxy);
    float *ha, *hb, *href, *gref;
    ha = new float[nxy], hb = new float[nxy], href = new float[nxy],
    gref = new float[nxy];
    {
        PROFILE_BLOCK("FILL MEMS HOST");
        fillData(ha, nxy);
        fillData(hb, nxy);
    }

    href = new float[nxy], gref = new float[nxy];
    float *da, *db, *dc;
    hipMalloc((void **)&da, nbytes);
    hipMalloc((void **)&db, nbytes);
    hipMalloc((void **)&dc, nbytes);
    {
        PROFILE_BLOCK("DATA HOST 2 DEVICE");
        hipMemcpy(da, ha, nbytes, hipMemcpyHostToDevice);
        hipMemcpy(db, hb, nbytes, hipMemcpyHostToDevice);
    }

    {
        PROFILE_BLOCK("HOST");
        sumMatrixHost(ha, hb, href, nx, ny);
    }

    {
        PROFILE_BLOCK("DEVICE");
        sumMatrixDevice(da, db, dc, nx, ny, 32, 32);
    }

    hipMemcpy(gref, dc, nbytes, hipMemcpyDeviceToHost);

    int cnt = check_diff(href, gref, nxy);
    std::cout << cnt << '\n';
    return 0;
}